#include "hip/hip_runtime.h"
#include "functors.h"
#include <thrust/iterator/transform_iterator.h>
#include <thrust/iterator/counting_iterator.h>
#include <thrust/generate.h>
#include <thrust/complex.h>
#include <thrust/functional.h>
#include <stdio.h>
#include <iostream>


int main() {
//    auto source = make_shared<File_source>("/mnt/raid/REC/900/TBS-880-940_25msps_gain_08_step_5e6.cf32");

    int N = 1024;
    int Nk = 64;
    int Nr = 64;
    int ROW = N * Nk;
    int SIZE = ROW * Nr;
    int num_rows = 1024;
    int num_columns = 1024;
    int buffer_size = num_rows * num_columns;
//    source->get_data_block(buffer_size);

    auto lfm_iter = tr::make_transform_iterator(tr::make_counting_iterator(0), ff::gen_lfm(N));

    std::cout << "alive" << std::endl;

    vcf packet(lfm_iter, lfm_iter + ROW);
    vcf chirp_fft(lfm_iter, lfm_iter + 200);
    tr::copy(lfm_iter, lfm_iter + N, chirp_fft.begin());

    vcf acc(N, cf(0));
    vcf conv(N, cf(0));

    auto fft_fun = cuda::fft(N);
    auto ifft_fun = cuda::ifft(N);

    fft_fun(chirp_fft, chirp_fft);
    tr::transform(chirp_fft.begin(), chirp_fft.end(), chirp_fft.begin(), ff::fn_conj());
//    tr::fill(packet.begin(), packet.end(), cf(1, 1));

//    суммируем по строкам
    for (int part = 0; part < Nk; part++) {
        tr::transform(packet.begin() + part * N,
                      packet.begin() + (part + 1) * N,
                      acc.data(),
                      acc.data(),
                      tr::plus<cf>());
    }

    // сворачиваем с оригиналом
    // сначала в conv кладем спектр от сигнала
    fft_fun(conv, acc);
    // перемножение с оригиналом
    tr::transform(conv.begin(), conv.end(), chirp_fft.begin(), conv.begin(), tr::multiplies<cf>());

    ifft_fun(conv);

    vf inreal(N);
    tr::transform(conv.begin(), conv.end(), inreal.begin(), ff::fn_abs());

    auto max = tr::max_element(inreal.begin(), inreal.end());
    auto pos = tr::distance(inreal.begin(), max);

    std::cout << "max_pos " << pos << std::endl;
}